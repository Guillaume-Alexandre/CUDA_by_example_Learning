//main

//####################################################################################################
// DOT PRODUCT USING ZEO MEMORY
//####################################################################################################

#ifndef ALL
#define ALL
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif // !ALL

#ifndef LIB
#define LIB
#include "lib\book.h"
#include "lib\cpu_anim.h"
#endif // !LIB

#include "kernel.cuh"
#include "functions.cuh"

int main(void) {
	//check device properties
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if (prop.canMapHostMemory != 1) {
		printf("Device cannot map memory.\n");
		return 0;
	}
	//tell the GPU to make Map memory
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
	//copy memory from CPU to GPU
	float elapsedTime = malloc_test(N);
	printf("Time using hipMalloc: %3.1f ms\n",
		elapsedTime);
	//Zerocopy memory
	elapsedTime = cuda_host_alloc_test(N);
	printf("Time using hipHostAlloc: %3.1f ms\n",
		elapsedTime);
	return 0;
}
