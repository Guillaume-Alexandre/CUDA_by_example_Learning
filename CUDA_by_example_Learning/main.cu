//main

#ifndef ALL
#define ALL
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif // !ALL

#ifndef LIB
#define LIB
#include "lib\book.h"
#include "lib\cpu_anim.h"
#endif // !LIB

#include "kernel.cuh"
#include "functions.cuh"

int main(void) {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no "
			"speed up from streams\n");
		return 0;
	}

	//Initialise timers
	hipEvent_t start, stop;
	float elapsedTime;
	// start the timers
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// initialize the stream we want to use for the application
	hipStream_t stream;
	HANDLE_ERROR(hipStreamCreate(&stream));

	
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;
	// allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a,
		N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b,
		N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c,
		N * sizeof(int)));

	// allocate page-locked memory, used to stream
	HANDLE_ERROR(hipHostAlloc((void**)&host_a,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&host_b,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&host_c,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	for (int i = 0; i<FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	// now loop over full data, in bite-sized chunks
	//It is important in the case where GPU has much less memory than the host
	for (int i = 0; i<FULL_DATA_SIZE; i += N) {
		// copy the locked memory to the device, async
		HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i,
			N * sizeof(int),
			hipMemcpyHostToDevice,
			stream));
		HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i,
			N * sizeof(int),
			hipMemcpyHostToDevice,
			stream));
		//compute
		kernel << <N / 256, 256, 0, stream >> >(dev_a, dev_b, dev_c);
		// copy the data from device to locked memory
		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c,
			N * sizeof(int),
			hipMemcpyDeviceToHost,
			stream));
	}
	// copy result chunk from locked to full buffer
	HANDLE_ERROR(hipStreamSynchronize(stream));

	//stop timers
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("Time taken: %3.1f ms\n", elapsedTime);


	// cleanup the streams and memory
	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
	//destroy stream
	HANDLE_ERROR(hipStreamDestroy(stream));
	return 0;
}
