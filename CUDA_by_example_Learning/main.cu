//kernel
#ifndef ALL
#define ALL
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif // !ALL

#ifndef LIB
#define LIB
#include "lib\book.h"
#include "lib\cpu_anim.h"
#endif // !LIB

#include "kernel.cuh"
#include "functions.cuh"

int main(void) {
	structure thing;

	function(thing.int_structure);

	return 0;

}
