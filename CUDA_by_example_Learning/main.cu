//main

#ifndef ALL
#define ALL
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif // !ALL

#ifndef LIB
#define LIB
#include "lib\book.h"
#endif // !LIB

#include "kernel.cuh"
#include "functions.cuh"

int main(void) {
	hipDeviceProp_t prop;
	int whichDevice;

	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);

	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipStream_t stream;
	hipStreamCreate(&stream);
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N) {
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

		kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}

	hipStreamSynchronize(stream);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Time taken: %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipStreamDestroy(stream);
	return 0;
}